#include "hip/hip_runtime.h"
/*
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <fstream>

#include <vector>
#include <list>
#include <set>
#include <utility>
#include <algorithm>

#include <percolated.h>

// includes, project
//#include <cutil_inline.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

// includes, kernels
#include <OSM_kernel.cu>

typedef float4 sph;
typedef thrust::device_vector<float4> d_sph_list;
typedef thrust::host_vector<float4>   h_sph_list;

using namespace std;

// For BGL connection algorithm
#include <boost/config.hpp>
#include <boost/graph/connected_components.hpp>
#include <boost/graph/adjacency_list.hpp>

// from BGL book p 201
using namespace boost;
typedef adjacency_list< vecS, vecS, undirectedS > UndirGraph;
typedef graph_traits< UndirGraph >::vertex_descriptor Vertex;
typedef graph_traits< UndirGraph >::out_edge_iterator OutEdgeIter;
typedef graph_traits< UndirGraph >::edge_descriptor EdgeDescriptor;


// For logging
#include <log.h>


#define cutilSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
#define cutilSafeThreadSync()        __cudaSafeThreadSync(__FILE__, __LINE__)

inline hipError_t cutilDeviceSynchronize()
{
#if CUDART_VERSION >= 4000
	return hipDeviceSynchronize();
#else
	return hipDeviceSynchronize();
#endif
}

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
		fprintf(stderr, "%s(%i) : cudaSafeCall() Runtime API error : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}

inline void __cudaSafeThreadSync( const char *file, const int line )
{
    hipError_t err = cutilDeviceSynchronize();
    if ( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : hipDeviceSynchronize() Runtime API error : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    srand ( time(NULL) );
    runTest( argc, argv);
    return 0;

//    cutilExit(argc, argv);
}

double randf()
{
    return (double)rand()/RAND_MAX;
}

float GetSphereRadius(float ret_r = -1)
{
    static float r = 0;
    if (ret_r > 0)
    {
        r = ret_r;
    }
    return r;
}

sph GenRndPoint(float3 dim_len)
{
    float4 result;
    result.x = randf() * dim_len.x;
    result.y = randf() * dim_len.y;
    result.z = randf() * dim_len.z;
    result.w = GetSphereRadius();
    return result;
}

const float max_overlapping = 0.4;

#define BLOCK_DIM 256


struct dist_gt
{
    sph curr;
    
    dist_gt(sph c)    {   curr = c;   }
    
    __host__ __device__
    bool operator()(const sph first, const sph second) const
    {
        float l1 = overlapping(first.w, curr.w, pnt_dist(first, curr));
        float l2 = overlapping(second.w, curr.w, pnt_dist(second, curr));
        
        //printf("L1 = %f, L2 = %f\n", l1, l2);
        return l1 > l2;
    }
};

struct dist_less
{   
    sph curr;

    dist_less(sph c)    {   curr = c;   }

    __host__ __device__
    bool operator()(const sph & first, const sph & second) const
    {
        float l1 = overlapping(first.w, curr.w, pnt_dist(first, curr));
        float l2 = overlapping(second.w, curr.w, pnt_dist(second, curr));

        //printf("L1 = %f, L2 = %f\n", l1, l2);
        return l1 < l2;
    }
};

float min_dist(float r1, float r2)
{
    float c = SQR(max_overlapping * (r1+r2));
    return 0.5 * (sqrt(4*SQR(r1) - c) + sqrt(4*SQR(r2) - c));
}

#define EPS 0.000001

bool in_space(const float3 & dim_len, const sph & pnt)
{
    return (0 <= pnt.x && pnt.x < dim_len.x &&
            0 <= pnt.y && pnt.y < dim_len.y &&
            0 <= pnt.z && pnt.z < dim_len.z);
}

bool move_pnt(const float3 & dim_len, const sph & center_sph, sph & moved_sph)
// returns true if point moved
// returns false if created new point
{
    float old_dist = pnt_dist(center_sph, moved_sph);
    if (old_dist < EPS)
    {
        moved_sph = GenRndPoint(dim_len);
        return false;
    }
    float r = min_dist(moved_sph.w, center_sph.w)/old_dist;
    moved_sph.x = (moved_sph.x - center_sph.x)*r + center_sph.x;
    moved_sph.y = (moved_sph.y - center_sph.y)*r + center_sph.y;
    moved_sph.z = (moved_sph.z - center_sph.z)*r + center_sph.z;
    if (!in_space(dim_len, moved_sph))
    {
        moved_sph = GenRndPoint(dim_len);
        return false;
    }
    return true;
}

ostream& operator<< (ostream& out, const float4& item )
{
    out << item.x << ", " << item.y << ", " << item.z << ", " << item.w;
    return out;
}

template <typename Iterator, typename BinaryPredicate>
Iterator my_max_element(Iterator begin, Iterator end, BinaryPredicate gt_op)
{
    Iterator result = begin;
    Iterator curr = result+1;
    while (curr != end)
    {
        if (gt_op(*curr, *result))
            result = curr;
        ++curr;
    }
    return result;
}

set<sph, dist_gt> * CollectNeighbours( h_sph_list::const_iterator start, h_sph_list::const_iterator stop, const sph curr)
// returns sorted set of neibours
{
    set<sph, dist_gt> * res = new set<sph, dist_gt>( dist_gt(curr) );
    const int cnt = stop - start;
// #pragma omp parallel for shared(cnt, res, curr)
    for (int idx = 0; idx < cnt; idx++)
    {
        sph curr_nei = *(start+idx);
        if (pnt_dist(curr_nei, curr) < 3 * curr.w)
        {
//            #pragma omp single
            res->insert(curr_nei);
        }
    }
    return res;
}

set<sph, dist_gt> * CollectNeighbours( sph * d_sph, h_sph_list & spheres, int cnt, const sph curr_sph)
// GPU version
{
//    printf("Start to CollectNeighbours\n");
    const int THREADS_PER_BLOCK = 256;
    if (cnt < THREADS_PER_BLOCK)
    {
        return CollectNeighbours(spheres.begin(), spheres.begin()+cnt, curr_sph);
    }
    int * d_results_idx = NULL;
    int * d_res_cnt = NULL;
    
    const int max_results = 1000;
    const int res_sz = max_results * sizeof(*d_results_idx);
    cutilSafeCall(hipMalloc((void **) &d_results_idx, res_sz));
    thrust::device_ptr<int> t_results_idx(d_results_idx);
    cutilSafeCall(hipMemset(d_results_idx, 0, res_sz));
    cutilSafeCall(hipMalloc((void **) &d_res_cnt, sizeof(*d_res_cnt)) );
    thrust::device_ptr<int> t_res_cnt(d_res_cnt);
    cutilSafeCall(hipMemset(d_res_cnt, 0, sizeof(*d_res_cnt)));
    
//    cout << "Curr d_res_cnt: " << t_res_cnt[0] << endl;
//    cout << "Curr d_curr_sph: " << t_curr_sph[0] << endl;
//    cout << "Curr sph: " << curr_sph << endl;
//    cout << "Curr d_cnt: " << t_cnt[0] << endl;
    
    
    int block = THREADS_PER_BLOCK;
    int grid = cnt / THREADS_PER_BLOCK;
    if (cnt % THREADS_PER_BLOCK != 0) grid += 1;
//    cout << grid << " " << cnt << " " << cnt / THREADS_PER_BLOCK << endl;
//    printf("Start GPU\n");
//    sph * d_sph = thrust::raw_pointer_cast(&d_spheres[0]);
//    printf("d_sph = %p\n", d_sph);
    nei_list<<<dim3(grid,1), dim3(block, 1, 1)>>>(d_sph, curr_sph, d_results_idx, d_res_cnt, cnt);
    cutilSafeCall(hipDeviceSynchronize());
    cutilSafeCall(hipGetLastError());
//    printf("GPU done\n");
    
    int results_cnt = t_res_cnt[0];
//    cout << "Res cnt = " << results_cnt << endl;
//    cutilSafeCall(hipMemcpy(&results_cnt, d_res_cnt, sizeof(int), hipMemcpyDeviceToHost));
//    cutilSafeCall(hipDeviceSynchronize());
    
    set<sph, dist_gt> * res = new set<sph, dist_gt>( dist_gt(curr_sph) );
    if (results_cnt != 0)   
    {
        int * results_idx = new int[results_cnt];
        try
        {
            thrust::copy(t_results_idx, t_results_idx+results_cnt, results_idx);
        }
        catch (thrust::system::system_error err)
        {
            cout << "Res_cnt = " << results_cnt << endl;
            cout << results_idx << endl;
            exit(100);
        }
//        cutilSafeCall(hipMemcpy(results_idx, d_results_idx, results_cnt*sizeof(int), hipMemcpyDeviceToHost));
//        cutilSafeCall(hipDeviceSynchronize());
        
//        printf("%d results copied\n", results_cnt);
        
        for (int idx = 0; idx < results_cnt; idx++)
        {
//            printf("sphere #%d\n", results_idx[idx]);
            res->insert(spheres[ results_idx[idx] ]);
        }
        
        delete [] results_idx;
    }
    hipFree(d_results_idx);
    hipFree(d_res_cnt);
    
    //printf("CollectNeighbours end\n");
    
//    set<sph, dist_gt> * tmp = CollectNeighbours(spheres.begin(), spheres.begin()+cnt, curr_sph);
//    cout << "CollectNeighbours results:\n";
//    set<sph, dist_gt>::const_iterator pnt;
//    cout << "CPU (" << tmp->size() << ")\n";
//    for (pnt = tmp->begin(); pnt != tmp->end(); ++pnt)
//    {
//        cout << *pnt << endl;
//    }
//    cout << "GPU (" << res->size() << ")\n";
//    for (pnt = res->begin(); pnt != res->end(); ++pnt)
//    {
//        cout << *pnt << endl;
//    }
//    delete tmp;
    return res;
}


int GenMaxPacked(const int max_cnt, const float3 dim_len, sph * spheres)
{
    h_sph_list h_spheres(max_cnt);
    int curr_cnt = 0;
    int max_holost = (int)(dim_len.x);
    int holost = 0;
    
    const int max_moves = 20;
    while (curr_cnt < max_cnt && holost++ < max_holost)
    {
        sph new_pnt = GenRndPoint(dim_len);
        //printf("New point (%i of %i): (%f, %f, %f)\n", curr_cnt, max_cnt, new_pnt.x, new_pnt.y, new_pnt.z);
        if (curr_cnt == 0) {
            h_spheres[curr_cnt] = new_pnt;
            cutilSafeCall(hipMemcpy(spheres+curr_cnt, &new_pnt, sizeof(sph), hipMemcpyHostToDevice));
            curr_cnt ++;
            holost = 0;
            continue;
        }
        bool add = false;
        bool maybe_add = false;
        int moves = 0;
        set<sph, dist_gt> * neigh = CollectNeighbours(spheres, h_spheres, curr_cnt, new_pnt);
        while (moves++ < max_moves)
        {
            if (neigh->empty())	{
                add = true;
                break;
            }
            sph over_sph = *(neigh->begin());
            if (is_overlapped(over_sph, new_pnt, max_overlapping)) {
                maybe_add = false;
                if (! move_pnt(dim_len, over_sph, new_pnt) )    {
                    delete neigh;
                    neigh = CollectNeighbours(spheres, h_spheres, curr_cnt, new_pnt);
                } else {
                    set<sph, dist_gt> * tmp = new set<sph, dist_gt>(dist_gt(new_pnt));
                    tmp->insert(neigh->begin(), neigh->end());
                    delete neigh;
                    neigh = tmp;
                    holost++;
                    moves = 0;
                }
            } else {
                if (!maybe_add) {
                    delete neigh;
                    neigh = CollectNeighbours(spheres, h_spheres, curr_cnt, new_pnt);
                    maybe_add = true;
                    continue;
                }
                add = true;
                break;
            }
        }
        if (add) {

            // test
//            for (int i = 0; i < curr_cnt; ++i)
//                if (is_overlapped(spheres[i], new_pnt, max_overlapping) )
//                {
//                    printf("Error!\n");
//                }
            h_spheres[curr_cnt] = new_pnt;
            cutilSafeCall(hipMemcpy(spheres+curr_cnt, &new_pnt, sizeof(sph), hipMemcpyHostToDevice));
            curr_cnt ++;
            holost = 0;
            if (curr_cnt % (max_cnt / 10) == 0)
            {
                time_t time_since_epoch;
                time( &time_since_epoch );
                tm *current_time = localtime( &time_since_epoch );
                
                cout << "Point #" << curr_cnt << " of " << max_cnt << ": " << asctime( current_time );
            }
        }
        delete neigh;
    }
    printf("Generated %d points\n", curr_cnt);
    return curr_cnt;
}

template <typename T>
class OutputItem
{
   public:
       explicit OutputItem( std::ofstream & stream )
                : stream_(&stream)
                {
                }

       void operator()( T const & item )
       {
           *stream_ << item.x << item.y << item.z << item.w;
       }

   private:
       std::ofstream * stream_;
};

void SaveToFile(const vector<sph> & spheres, const char * filename)
{
    FILE * outFile = fopen(filename, "wb");
    
    for (int i = 0; i < spheres.size(); ++i)
    {
        fwrite(&(spheres[i]), sizeof(spheres[i].x), 4, outFile);
    }
    
    fclose(outFile);
    printf("%d spheres saved to file %s\n", spheres.size(), filename);
}

vector<sph> * LoadFromFile( const char * filename)
{
    FILE * inFile = fopen(filename, "rb");
    sph curr_pnt;
    vector<sph> * tmp = new vector<sph>();
    while(fread(&curr_pnt, sizeof(curr_pnt.x), 4, inFile))
        tmp->push_back( curr_pnt );
    return tmp;
}

template <typename OutputType>
void print(OutputType v)
{
    cout << v << " ";
}

template <typename OutputType>
void println(OutputType v)
{
    cout << v << endl;
}

struct BorderIndex
{
    BorderIndex(const vector<sph> & spheres, const float3 sz):
    m_spheres(&spheres), m_sz(sz)
    {
    }
    
    vector<int> * operator()(int idx)
    {
        vector<int> * res = new vector<int>;
        sph curr_sph = m_spheres->at(idx);
        if (curr_sph.x-curr_sph.w < 0)
            res->push_back(0);
        if (curr_sph.x+curr_sph.w > m_sz.x)
            res->push_back(1);
        if (curr_sph.y-curr_sph.w < 0)
            res->push_back(2);
        if (curr_sph.y+curr_sph.w > m_sz.y)
       	    res->push_back(3);
        if (curr_sph.z-curr_sph.w < 0)
            res->push_back(4);
        if (curr_sph.z+curr_sph.w > m_sz.z)
       	    res->push_back(5);
       	return res;
    }
private:
    const vector<sph> * m_spheres;
    float3 m_sz;
};

//vector<vector<sph> > * PercolatedClusters( const list<sph> & spheres, const float3 sz )
//{
//    std::vector<int> clusters(num_vertices(vg));
//    int num = 
//    connected_components(vg, make_iterator_property_map(clusters.begin(), get(vertex_index, vg), clusters[0]));
//
//    set<int> * borders = new set<int>[6];
//    // find all spheres on the borders
//    // and save cluster numbers
//    int sph_idx = 0;
//    for (it1 = spheres.begin(); it1 != spheres.end(); ++it1, ++sph_idx)
//    {
//        sph curr_sph = *it1;
//        if (curr_sph.x-curr_sph.w < 0)
//            borders[0].insert(clusters[sph_idx]);
//        if (curr_sph.x+curr_sph.w > sz.x)
//            borders[1].insert(clusters[sph_idx]);
//        if (curr_sph.y-curr_sph.w < 0)
//            borders[2].insert(clusters[sph_idx]);
//        if (curr_sph.y+curr_sph.w > sz.y)
//       	    borders[3].insert(clusters[sph_idx]);
//        if (curr_sph.z-curr_sph.w < 0)
//            borders[4].insert(clusters[sph_idx]);
//        if (curr_sph.z+curr_sph.w > sz.z)
//       	    borders[5].insert(clusters[sph_idx]);
//    }
//    // find intersection between borders
//    int min_size = borders[0].size();
//    for (int dim = 1; dim < 6; ++dim)
//    {
//        if (borders[dim].size() < min_size)
//            min_size = borders[dim].size();
//    }
//    if (min_size == 0)
//    {
//        printf("Not percolate\n");
//        delete [] borders;
//        return NULL;
//    }
//    vector<int> * perc_clusters = new vector<int>(borders[0].begin(), borders[0].end());
//    vector<int>::iterator last_it = perc_clusters->end();
//    vector<int> tmp(perc_clusters->size());
//    for (int dim = 1; dim < 6; ++dim)
//    {
//        vector<int>::iterator it = set_intersection(perc_clusters->begin(), last_it, borders[dim].begin(), borders[dim].end(), tmp.begin());
//        if (it - tmp.begin() == 0)
//        {
//            printf("Non perc [%d]\n", dim);
//            delete [] borders;
//            return NULL;
//        }
//        last_it = copy(tmp.begin(), it, perc_clusters->begin());
//    }
//    perc_clusters->resize(last_it-perc_clusters->begin());
//    
//    vector<vector <sph> > * res = new vector<vector <sph> >(perc_clusters->size());
//    
//    int clust_idx = 0;
//    for (vector<int>::iterator it = perc_clusters->begin(); 
//         it != perc_clusters->end(); ++it, ++clust_idx)
//    {
//        it1 = spheres.begin();
//        for (vector<int>::iterator cl_it = clusters.begin(); cl_it != clusters.end(); ++cl_it, ++it1)
//            if (*cl_it == *it)
//                res->at(clust_idx).push_back(*it1);
//    }
//    printf("Percolated clusters:\n");
//    for (vector<vector<sph> >::iterator it = res->begin(); it != res->end(); ++it)
//        println(it->size());
//    
//    return res;
//}

double Volume(double radius)
{
    return (4.0/3.0) * 3.14159 * (radius*radius*radius);
}

double CalcVolume(const vector<sph> & spheres, const vector<int> & indicies)
{
    double res = 0;
    vector<int>::const_iterator it = indicies.begin();
    while(it != indicies.end()) 
    {
        res += Volume(spheres[*it].w);
        ++it;
    }
    return res;
}

struct Adjust
{
    Adjust(const vector<sph> & spheres, float max_over):
    m_max_overlapping(max_over),
    m_spheres(spheres)
    {
    }
    bool operator()(int idx1, int idx2)
    {
        return slightly_overlap(m_spheres[idx1], m_spheres[idx2], m_max_overlapping);
    }
private:
    vector<sph> m_spheres;
    float m_max_overlapping;
};

vector<sph> * ConvertIndToSph(const vector<sph> & spheres, const vector<int> & indicies)
{
    vector<sph> * res = new vector<sph>(indicies.size());
    for (int idx = 0; idx < indicies.size(); ++idx)
    {
        res->at(idx) = spheres[indicies[idx]];
    }
    return res;
}

UndirGraph * ConvertSphToGraph(const vector<sph> & spheres)
{
    d_sph_list d_sph(spheres.begin(), spheres.end());
    
    const int THREADS_CNT = 256;
    int grid_dim = spheres.size()/THREADS_CNT;
    if (spheres.size()/THREADS_CNT != 0) grid_dim += 1;
    dim3 grid(grid_dim, 1);
    dim3 block(THREADS_CNT, 1, 1);
    
    const int res_cnt = 100 + 1;
    
    int * d_results = NULL;
    int * h_results = new int[res_cnt];
    sph * d_spheres_ptr = thrust::raw_pointer_cast(&d_sph[0]);
    const size_t res_sz = res_cnt * sizeof(int); // max 100 results + 0th element – res_cnt
    
    hipMalloc((void **) &d_results, res_sz);
    
    UndirGraph * vg = new UndirGraph(spheres.size());
    int curr_vertex;
    for (curr_vertex = 0; curr_vertex < spheres.size(); ++curr_vertex)
    {
        hipMemset(d_results, 0, res_sz);
        slight_nei_list<<<grid, block>>>(d_spheres_ptr, curr_vertex, spheres.size(), max_overlapping, d_results);
        hipDeviceSynchronize();
        hipMemcpy(h_results, d_results, res_sz, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        if (h_results[0] >= res_cnt)
        {
            printf("Too much results!\n");
            exit(199);
        }
        for (int adj_vertex = 0; adj_vertex < h_results[0]; ++adj_vertex)
        {
            add_edge(curr_vertex, h_results[adj_vertex+1], *vg);
        }
    }
    return vg;
}

vector<sph> * RemovePoints( const vector<sph> & spheres, const float3 sz, const double min_volume )
{
    printf("Start to convert points... ");
    UndirGraph * vg = ConvertSphToGraph(spheres);
    printf("Done\n");
    Percolation<Adjust, BorderIndex > perc(*vg, spheres.size(), Adjust(spheres, max_overlapping), BorderIndex(spheres, sz));
    delete vg;
    
    if (!perc.IsPercolated())
    {
        printf("Can\'t remove points!\n");
        return NULL;
    }
    
    // choose biggest cluster:
    double max_cluster_size = CalcVolume(spheres, perc.GetPercClusterItems(0));
    int max_cluster_idx = 0;
    for (int i = 1; i < perc.GetPercClustersCnt(); ++i)
    {
        double vol = CalcVolume(spheres, perc.GetPercClusterItems(i));
        if (vol > max_cluster_size)
        {
            max_cluster_size = vol;
            max_cluster_idx = i;
        }
    }
    if (max_cluster_size < min_volume)
    {
        printf("Percolated cluster too small\n");
        return NULL;
    }
    perc.OnlyPerc(max_cluster_idx);
    
    printf("Start deleting operations\n");
    
    while(1)
    {
        int del_idx = perc.TestRandomVertex();
        if (del_idx == -1)
        {
            log_it("Nope..");
            continue;
        } else if (del_idx == -2) {
            log_it("Spheres goes to end.");
            return NULL; // TODO: return final cluster
        }
        
        // choose biggest cluster:
        double max_cluster_size = CalcVolume(spheres, perc.GetPercClusterItems(0));
        int max_cluster_idx = 0;
        for (int i = 1; i < perc.GetPercClustersCnt(); ++i)
        {
            double vol = CalcVolume(spheres, perc.GetPercClusterItems(i));
            if (vol > max_cluster_size)
            {
                max_cluster_size = vol;
                max_cluster_idx = i;
            }
        }
        printf("Biggest cluster have volume: %f\n", max_cluster_size);
        // +- 1% of min_volume is acceptable
        if (max_cluster_size < 0.99*min_volume)
        {
            perc.RestoreState();
            printf("Cluster too small\n");
            continue;
        }
        if (max_cluster_size < 1.01*min_volume)
        {
            vector<sph> *res = ConvertIndToSph(spheres, perc.GetPercClusterItems(max_cluster_idx));
            return res;
        }
        perc.StopSaving();
        perc.OnlyPerc(max_cluster_idx);
        printf("Current volume = %f, must be %f\n", max_cluster_size, min_volume);
    }
    // never come here
}

void
runTest( int argc, char** argv) 
{
    const float dim_sz = 500.0f;
    const double e_max = 0.3f;
    const float r = 3.0;
    GetSphereRadius(r);
    
    const float3 sz = make_float3(dim_sz,dim_sz,dim_sz);
    const double vol = sz.x * sz.y * sz.z;
    const double vol_sph = Volume(r);
    const int max_cnt =(int) (vol / vol_sph * (1.0-e_max));
    
    unsigned pref_gpu = 0;
    if (argc > 1)
    {
        int res = sscanf(argv[1], "%u", &pref_gpu);
        if (res == EOF)
        {
            printf("Using: %s [gpu_number]", argv[0]);
            exit(10);
        }
    }
    cutilSafeCall(hipSetDevice((int)pref_gpu));
    cout << "GPU#" << pref_gpu << endl;

    cout << "Start\n";
    
//    cout << "Loading\n";
//    vector<sph> * v_spheres = LoadFromFile("max_500_r25_gpu.dat");
//    d_sph_list d_spheres(max_cnt);


    sph * d_spheres_raw = NULL;
    cutilSafeCall(hipMalloc((void **) &d_spheres_raw, max_cnt*sizeof(sph)));
    cutilSafeCall(hipMemset(d_spheres_raw, 0, max_cnt*sizeof(sph)));
    int cnt = GenMaxPacked(max_cnt, sz, d_spheres_raw);
    thrust::device_ptr<sph> d_spheres(d_spheres_raw);
    h_sph_list spheres(d_spheres, d_spheres + cnt);
    hipFree(d_spheres_raw);
//    cout << "Test passed. Saving\n";
    vector<sph> * v_spheres = new vector<sph>(spheres.begin(), spheres.end());
    SaveToFile(*v_spheres, "max_500_r30_gpu_2.dat");
    
    // test
//    cout << "Start test\n";
//    for (int idx1 = 0; idx1 < spheres.size(); ++idx1)
//    {
//        for (int idx2 = idx1+1; idx2 < spheres.size(); ++idx2)
//        {
//            if (is_overlapped(spheres[idx1], spheres[idx2], max_overlapping))
//            {
//                cout << "Test failed! 1:" << spheres[idx1] << " 2: " << spheres[idx2] << endl;
//                exit(200);
//            }
//        }
//    }
    double need_e = 1.0-0.1/2.2;
    double need_vol = vol*(1-need_e);
    vector<sph> * res = RemovePoints(*v_spheres, sz, need_vol);

//    for (int idx1 = 0; idx1 < spheres.size(); ++idx1)
//    {
//        for (int idx2 = idx1+1; idx2 < spheres.size(); ++idx2)
//        {
//            if (is_overlapped(res[0][idx1], res[0][idx2], max_overlapping))
//            {
//                cout << "Test failed! 1: " << res[0][idx1] << " 2: " << res[0][idx2] << endl;
//               // return;
//            }
//        }
//    }
    //h_sph_list h_spheres(spheres.begin(), spheres.begin() + cnt);
    //
    SaveToFile( *res, "res_500_r30_90_gpu_2.dat");
    delete res;
    
    //cout << "Done. Points: " << cnt << " of " << max_cnt
    //<< ". E = " << (1 - vol_sph * cnt / vol) << endl;
}
